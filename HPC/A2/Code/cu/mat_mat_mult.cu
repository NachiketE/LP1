

#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__
void matrixMultiplication(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int sum=0;
    if(col<k && row<m) {
      for(int j=0;j<n;j++)
      {
          sum += a[row*n+j] * b[j*k+col];
      }
      c[k*row+col]=sum;
    }
    
}

void init_result(int *a, int m, int k) {
    for(int i=0; i<m; i++) {
      for(int j=0; j<k; j++) {
        a[i*k + j] = 0;
      }
    }
}

void init_matrix(int *a, int n, int m) {
  srand((unsigned)time(0));
    for(int i=0; i<n; i++) {
      for(int j=0; j<m; j++) {
        a[i*m + j] = rand()%10 + 1;
      }
    }
}

void print_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
      for(int j=0; j<m; j++) {
        cout<<"  "<<a[i*m + j];
      }
      cout<<endl;
    }
    cout<<endl;
}

int main()
{
    
    int *a,*b,*c;
    int *a_dev,*b_dev,*c_dev;
    int m=5, n=4, k=3;
    
    a = new int[m*n];
    b = new int[n*k];
    c = new int[m*k];
    
    init_matrix(a, m, n);
    init_matrix(b, n ,k);
    init_result(c, m, k);
    
    cout<<"Initial matrix : "<<endl;
    
    print_matrix(a, m, n);
    print_matrix(b, n, k);
   // print_matrix(c, m, k);
    
    hipMalloc(&a_dev, sizeof(int)*m*n);
    hipMalloc(&b_dev, sizeof(int)*n*k);
    hipMalloc(&c_dev, sizeof(int)*m*k);
       
    hipMemcpy(a_dev, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, sizeof(int)*n*k, hipMemcpyHostToDevice);
    
    dim3 dimGrid(1,1);
    dim3 dimBlock(16,16);
    
    matrixMultiplication<<<dimGrid, dimBlock>>>(a_dev,b_dev,c_dev, m, n, k);
     
    hipMemcpy(c, c_dev, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    
    cout<<"Result : "<<endl;
    print_matrix(c, m, k);
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    
    delete[] a;
    delete[] b;
    delete[] c;
    
    return 0;
}
