
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__
void matrixVector(int *vec, int *mat, int *result, int n, int m)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int sum=0;
    
    if(tid <= n) {
        for(int i=0; i<n; i++) {
            sum += vec[i]*mat[(i*m) + tid];
        }
        result[tid] = sum;
    }
}

void init_array(int *a, int n) {
    for(int i=0; i<n; i++)
      a[i] = rand()%n + 1;
}

void init_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<m; j++) {
            a[i*m + j] = rand()%n + 1;
        }
    }
}

void print_array(int *a, int n) {
    for(int i=0; i<n; i++) {
        cout<<"  "<<a[i];
    }
    cout<<endl;
}

void print_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<m; j++)
          cout<<"  "<<a[i*m + j];
        cout<<endl;
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    
    int n = 3;
    int m = 4;
    
    a = new int[n];
    b = new int[n*m];
    c = new int[m];
    
    init_array(a, n);
    init_matrix(b, n, m);
        
    cout<<"Initial array : "<<endl;
    print_array(a, n);
    cout<<"Initial matrix : "<<endl;
    print_matrix(b, n, m);
    cout<<"Initial resultant array : "<<endl;
    print_array(c, m);
    cout<<endl;
    
    hipMalloc(&a_dev, sizeof(int)*n);
    hipMalloc(&b_dev, sizeof(int)*n*m);
    hipMalloc(&c_dev, sizeof(int)*m);
    
    hipMemcpy(a_dev, a, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, sizeof(int)*n*m, hipMemcpyHostToDevice);
    
    matrixVector<<<m/256+1, 256>>>(a_dev, b_dev, c_dev, n, m);
    
    hipMemcpy(c, c_dev, sizeof(int)*m, hipMemcpyDeviceToHost);
    
    cout<<"Results : "<<endl;
    print_array(c, m);
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    
    delete[] a;
    delete[] b;
    delete[] c;
    
    return 0;
}


